#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"

//#define THREAD_PER_BLOCK 256
//估计这种warp和shared在老的gpu上面会很有成效，但是在turing后的GPU，nvcc编译器优化了很多
//cpu
int filter(int *dst, int *src, int n) {
  int nres = 0;
  for (int i = 0; i < n; i++)
    if (src[i] > 0)
      dst[nres++] = src[i];
  // return the number of elements copied
  return nres;
}
  
//数据量为256000000时，latency=14.37ms
//cuda naive kernel
__global__ void filter_k(int *dst, int *nres, int *src, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < n && src[i] > 0){
    dst[atomicAdd(nres, 1)] = src[i];
    // dst[*nres++] = src[i];
  }
}

//数据量为256000000时，latency=13.86ms
//block level, use block level atomics based on shared memory
__global__ 
void filter_shared_k(int *dst, int *nres, const int* src, int n) {
  __shared__ int l_n;
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  int total_thread_num = blockDim.x * gridDim.x;

  for (int i = gtid; i < n; i += total_thread_num) {
    // use first thread to zero the counter
    if (threadIdx.x == 0)
      l_n = 0;
    __syncthreads();

    // 每个block内部，大于0的数量(l_n)和每个大于0的thread offset(pos)
    int d, pos;

    if(i < n) {
      d = src[i];
      if(d > 0)
        //pos: src[thread]>0的thread在当前block的index
        pos = atomicAdd(&l_n, 1);
    }
    __syncthreads();

    //每个block选出tid=0作为leader
    //leader把每个block的大于0的数量l_n累加到 the global counter(nres)
    if(threadIdx.x == 0)
      l_n = atomicAdd(nres, l_n);
    __syncthreads();

    //write & store
    if(i < n && d > 0) {
    //pos: src[thread]>0的thread在当前block的index
    //l_n: 在当前block的前面几个block的所有src>0的个数
    //pos + l_n：当前thread的全局offset
      pos += l_n; 
      dst[pos] = d;
    }
    __syncthreads();
  }
}

//数据量为256000000时，latency=13.79ms
//warp level, use warp-aggregated atomic
__device__ int atomicAggInc(int *ctr) {
  // active 1,idle 0
  unsigned int active = __activemask(); 
  int leader = 0;
  int change = __popc(active);//warp mask中为1的数量
  int lane_mask_lt;
  asm("mov.u32 %0, %%lanemask_lt;" : "=r"(lane_mask_lt));
  unsigned int rank = __popc(active & lane_mask_lt);//比当前线程id小且值为1的mask之和
  int warp_res;
  if(rank == 0)//leader thread of every warp
    warp_res = atomicAdd(ctr, change);//compute global offset of warp
  warp_res = __shfl_sync(active, warp_res, leader);//broadcast to every thread
  return warp_res + rank;
}

__global__ void filter_warp_k(int *dst, int *nres, const int *src, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i >= n)
    return;
  if(src[i] > 0)
    dst[atomicAggInc(nres)] = src[i];
}


bool CheckResult(int *out, int groudtruth, int n){
    //for (int i = 0; i < n; i++){
    if (*out != groudtruth) {
        return false;
    }
    //}
    return true;
}

bool CheckResultCarefully(int *out_d, int *out_h, int nres, int groudtruth){
  if (nres != groudtruth){
    return false;
  }
  for (int i = 0; i < groudtruth; i++){
    if (out_d[i] != out_h[i]){
      return false;
    }
  } 
  return true;
} 

int main(){
    float milliseconds = 0;
    int N = 256000000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);

    int *src_h = (int *)malloc(N * sizeof(int));
    int *dst_h = (int *)malloc(N * sizeof(int));
    // int *dst_d = (int *)malloc(N * sizeof(int));
    int *nres_h = (int *)malloc(1 * sizeof(int));

    int *dst, *nres;
    int *src;
    hipMalloc((void **)&src, N * sizeof(int));
    hipMalloc((void **)&dst, N * sizeof(int));
    hipMalloc((void **)&nres, 1 * sizeof(int));
    
    // for(int i = 0; i < N; i++){
    //     src_h[i] = rand() % 17 - 8;
    // }

    // int groudtruth = 0;
    // for(int j = 0; j < N; j++){
    //     if (src_h[j] > 0) {
    //         groudtruth += 1;
    //     }
    // }

    int groudtruth = 0;
    // [m,n] 内随机数 rand() % (n - m + 1) + m
    for(int i = 0; i < N; i++){
        src_h[i] = rand() % 17 - 8;
        if (src_h[i] > 0){
          groudtruth += 1;
        }
    }

    auto cpu_start = std::chrono::steady_clock::now();
    filter(dst_h,src_h,N);
    auto cpu_end = std::chrono::steady_clock::now();
    double dr_ns=std::chrono::duration<double,std::nano>(cpu_end-cpu_start).count(); 
    printf("CPU %lf ms\n",dr_ns / 1000.0f / 1000.0f);

    hipMemcpy(src, src_h, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // filter_k<<<Grid, Block>>>(dst, nres, src, N);
    // filter_shared_k<<<Grid, Block>>>(dst, nres, src, N);
    filter_warp_k<<<Grid, Block>>>(dst, nres, src, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);


    hipMemcpy(nres_h, nres, 1 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(nres_h, groudtruth, N);

    // hipMemcpy(dst_d,dst,N * sizeof(int),hipMemcpyDeviceToHost);
    // bool is_right = CheckResultCarefully(dst_d, dst_h, *nres_h, groudtruth);

    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        printf("Count curr %d ,but %d\n",groudtruth,*nres_h);
    }
    printf("filter_k latency = %f ms\n", milliseconds);    

    hipFree(src);
    hipFree(dst);
    hipFree(nres);
    free(src_h);
    free(dst_h);
    free(nres_h);
}
