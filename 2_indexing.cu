#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void sum(float *x)
{
    int block_id = blockIdx.x;
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    printf("current block=%d, thread id in current block =%d, global thread id=%d\n", block_id, local_tid, global_tid);
    x[global_tid] += 1;
}

int main(){
    int N = 32;
    int nbytes = N * sizeof(float);
    // *dx 指向显存, *hx 内存
    float *dx, *hx;
    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);//思考为什么要用二级指针
    /* allocate CPU mem */
    hx = (float*) malloc(nbytes);
    /* init host data */
    printf("hx original: \n");
    for (int i = 0; i < N; i++) {
        hx[i] = i;
        printf("%g\n", hx[i]);
    }
    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    /* launch GPU kernel */
    sum<<<1, N>>>(dx);
    /* copy data from GPU */
    hipMemcpy(hx, dx, nbytes, hipMemcpyDeviceToHost);
    printf("hx current: \n");
    for (int i = 0; i < N; i++) {
        printf("%g\n", hx[i]);
    }
    hipFree(dx);
    free(hx);
    return 0;
}
