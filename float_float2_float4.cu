#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

#define BLOCKSIZE 256

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/********************/
/* ADD_FLOAT KERNEL */
/********************/
__global__ void add_float(float *d_a, float *d_b, float *d_c, unsigned int N) {

    const int tid = 4 * threadIdx.x + blockIdx.x * (4 * blockDim.x);

    if (tid < N) {

        float a1 = d_a[tid];
        float b1 = d_b[tid];

        float a2 = d_a[tid+1];
        float b2 = d_b[tid+1];

        float a3 = d_a[tid+2];
        float b3 = d_b[tid+2];

        float a4 = d_a[tid+3];
        float b4 = d_b[tid+3];

        float c1 = a1 + b1;
        float c2 = a2 + b2;
        float c3 = a3 + b3;
        float c4 = a4 + b4;

        d_c[tid] = c1;
        d_c[tid+1] = c2;
        d_c[tid+2] = c3;
        d_c[tid+3] = c4;

        //if ((tid < 1800) && (tid > 1790)) {
            //printf("%i %i %i %f %f %f\n", tid, threadIdx.x, blockIdx.x, a1, b1, c1);
            //printf("%i %i %i %f %f %f\n", tid+1, threadIdx.x, blockIdx.x, a2, b2, c2);
            //printf("%i %i %i %f %f %f\n", tid+2, threadIdx.x, blockIdx.x, a3, b3, c3);
            //printf("%i %i %i %f %f %f\n", tid+3, threadIdx.x, blockIdx.x, a4, b4, c4);
        //}

    }

}

/*********************/
/* ADD_FLOAT2 KERNEL */
/*********************/
__global__ void add_float2(float2 *d_a, float2 *d_b, float2 *d_c, unsigned int N) {

    const int tid = 2 * threadIdx.x + blockIdx.x * (2 * blockDim.x);

    if (tid < N) {

        float2 a1 = d_a[tid];
        float2 b1 = d_b[tid];

        float2 a2 = d_a[tid+1];
        float2 b2 = d_b[tid+1];

        float2 c1;
        c1.x = a1.x + b1.x;
        c1.y = a1.y + b1.y;

        float2 c2;
        c2.x = a2.x + b2.x;
        c2.y = a2.y + b2.y;

        d_c[tid] = c1;
        d_c[tid+1] = c2;

    }

}

/*********************/
/* ADD_FLOAT4 KERNEL */
/*********************/
__global__ void add_float4(float4 *d_a, float4 *d_b, float4 *d_c, unsigned int N) {

    const int tid = 1 * threadIdx.x + blockIdx.x * (1 * blockDim.x);

    if (tid < N/4) {

        float4 a1 = d_a[tid];
        float4 b1 = d_b[tid];

        float4 c1;
        c1.x = a1.x + b1.x;
        c1.y = a1.y + b1.y;
        c1.z = a1.z + b1.z;
        c1.w = a1.w + b1.w;

        d_c[tid] = c1;

    }

}

/********/
/* MAIN */
/********/
int main() {

    const int N = 8*100000000; //10M

    const float a = 3.f;
    const float b = 5.f;
    float *res = new float[N];
    for (int i = 0;i < N;i++){
        res[i] = 8.f;
    }

    // --- float

    thrust::device_vector<float> d_A(N, a);
    thrust::device_vector<float> d_B(N, b);
    thrust::device_vector<float> d_C(N);

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    add_float<<<iDivUp(N/4, BLOCKSIZE), BLOCKSIZE>>>(thrust::raw_pointer_cast(d_A.data()), thrust::raw_pointer_cast(d_B.data()), thrust::raw_pointer_cast(d_C.data()), N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("float Elapsed time:  %3.1f ms \n", time); gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    thrust::host_vector<float> h_float = d_C;
    for (int i=0; i<N; i++) {
        if (h_float[i] != res[i]) {
            printf("Error for add_float at %i: result is %f\n",i, h_float[i]);
            return -1;
        }
    }

    // --- float2
    
    // thrust::device_vector<float> d_A2(N, a);
    // thrust::device_vector<float> d_B2(N, b);
    // thrust::device_vector<float> d_C2(N);

    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    add_float2<<<iDivUp(N/4, BLOCKSIZE), BLOCKSIZE>>>((float2*)thrust::raw_pointer_cast(d_A.data()), (float2*)thrust::raw_pointer_cast(d_B.data()), (float2*)thrust::raw_pointer_cast(d_C.data()), N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("float2 Elapsed time:  %3.1f ms \n", time); gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    thrust::host_vector<float> h_float2 = d_C;
    for (int i=0; i<N; i++) {
        if (h_float2[i] != res[i]) {
            printf("Error for add_float2 at %i: result is %f\n",i, h_float2[i]);
            return -1;
        }
    }

    // --- float4

    // thrust::device_vector<float> d_A4(N, a);
    // thrust::device_vector<float> d_B4(N, b);
    // thrust::device_vector<float> d_C4(N);

    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    add_float4<<<iDivUp(N/4, BLOCKSIZE), BLOCKSIZE>>>((float4*)thrust::raw_pointer_cast(d_A.data()), (float4*)thrust::raw_pointer_cast(d_B.data()), (float4*)thrust::raw_pointer_cast(d_C.data()), N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("float4 Elapsed time:  %3.1f ms \n", time); gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    thrust::host_vector<float> h_float4 = d_C;
    for (int i=0; i<N; i++) {
        if (h_float4[i] != res[i]) {
            printf("Error for add_float4 at %i: result is %f\n",i, h_float4[i]);
            return -1;
        }
    }

    return 0;
}